#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
//define GET_TIME
#ifndef _TIMER_H_
#define _TIMER_H_
typedef struct
{
	double Y;
	double Y1;
	double Y2;
}interp_t;
#include <time.h>

#define GET_TIME(now){ \
	double t; \
	t=clock(); \
	now =t/CLOCKS_PER_SEC ;\
}
#endif

#define CudaBlckNum 32
#define CudaThrdNum 32
#define NumOfNodes 320
#define NumOfElems 578
#define Delt (0.01)

typedef struct
{
	int NumEle;
	int EleID[10];
	int EleOrd[10];
	int Type;
	float Jext;
	float Jeddy = 0.0;
	float A;// not in use-> relpaced by float *d_x outside of struct
	float A1;// not in use-> relpaced by float *d_x outside of struct
	float TLM_Eq_Jsource = 0.0;

}FEMNode;
typedef struct
{
	int I, J, K;
	int Type;
	float Ve;
	float Kc[3][3];//Kc ��G01 G12����ϵġ�ע��ֻ��������һ��
	float Dp[3][3];
	float MatrixMultiVecBuff[3];
	float Dp1;
	float Dp2;
	//TLM vars
	float Area;
	float Vr[3] = { 0.0,0.0,0.0 };// Vr[0]=Vr01,[1]=12,[2]=20
	float Vi[3] = { 0.0,0.0,0.0 };
	// Vr , Vi is with respect to nonlinear element resistors
	float Ieq[3];// equvlt currents by Vr from nonlinear resistors
}FEMElem;

//FEM vars
//------------FEM vars host
FEMNode MyNode[NumOfNodes];
FEMElem MyElem[NumOfElems];
int NumNodes, NumElems;
//------------FEM vars device
FEMNode *d_MyNode;
FEMElem *d_MyElem;
int *d_NumNodes, *d_NumElems;
int *d_WannaUpdateVe;
float *d_yinuonuo, *d_anuonuo;//dummys watchers for debug


//PCG deviece vars solving Ax=b
typedef struct
{
	float r1, r0, z1, z0, b, Ap, PreCondi;
}PcgVec;
PcgVec *d_PcgVec;// PCG solving vector var package floating clouds 
float *d_x;// x= A,the mag potential on each nodes. cmmunication vects
float *d_p;// cmmunication vects
float *d_buffer1, *d_buffer2; //reduction buffers
float *d_alpha, *d_beta, *d_pap, *d_rz1, *d_rz0; //floating clouds PCG steping vars pap=p'*A*p; rz1=r1'*z1 ...

// prepare data and memory control
void LoadMeshInfoAndPrepareFEM()
{
	FILE* ip;
	int i, flag = 0;
	char filename[50];
	
	char line[50];

	sprintf(filename, "Untitled.mphtxt");


	//load mesh connections and coord
	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}
	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# number of mesh points") != NULL)
		{
			sscanf(line, "%d", &(NumNodes));
		}
	}
	fclose(ip);

	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}
	printf("Num_nodes = %d \n", NumNodes);

	double *X, *Y;
	X = (double *)malloc(NumNodes * sizeof(double));
	Y = (double *)malloc(NumNodes * sizeof(double));
	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# Mesh point coordinates") != NULL)
		{
			for (i = 0; i < NumNodes; i++)
			{
				fgets(line, sizeof(line), ip);
				sscanf(line, "%lf %lf\n", &X[i], &Y[i]);
			}
		}
	}
	fclose(ip);

	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}
	flag = 0;
	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# number of elements") != NULL)
		{
			flag = flag + 1;
			if (flag == 3)
			{
				sscanf(line, "%d", &(NumElems));
				fgets(line, sizeof(line), ip);
				printf("Num_elems = %d \n", NumElems);
				for (i = 0; i < NumElems; i++)
				{
					fgets(line, sizeof(line), ip);
					sscanf(line, "%d %d %d\n", &(MyElem[i].I), &(MyElem[i].J), &(MyElem[i].K));
				}
			}
		}
	}
	fclose(ip);

	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}
	flag = 0;
	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# number of geometric entity indices") != NULL)
		{
			flag = flag + 1;
			if (flag == 3)
			{
				sscanf(line, "%d", &(NumElems));
				fgets(line, sizeof(line), ip);
				//printf("Num_elems = %d \n", NumElems);
				for (i = 0; i < NumElems; i++)
				{
					fgets(line, sizeof(line), ip);
					sscanf(line, "%d\n", &(MyElem[i].Type));
				}
			}
		}
	}
	fclose(ip);
	//load mesh connections and coord finished

	//link NDD topology according to mesh connecitons
	for (i = 0; i < NumNodes; i++)
		MyNode[i].NumEle = 0;
	int nownode;
	for (i = 0; i < NumElems; i++)
	{
		nownode = MyElem[i].I;
		MyNode[nownode].EleID[MyNode[nownode].NumEle] = i;
		MyNode[nownode].EleOrd[MyNode[nownode].NumEle] = 0;
		MyNode[nownode].NumEle += 1;
		nownode = MyElem[i].J;
		MyNode[nownode].EleID[MyNode[nownode].NumEle] = i;
		MyNode[nownode].EleOrd[MyNode[nownode].NumEle] = 1;
		MyNode[nownode].NumEle += 1;
		nownode = MyElem[i].K;
		MyNode[nownode].EleID[MyNode[nownode].NumEle] = i;
		MyNode[nownode].EleOrd[MyNode[nownode].NumEle] = 2;
		MyNode[nownode].NumEle += 1;
	}
	//link NDD topology according to mesh connecitons finshed

	//mark boundary nodes and set Jext to 0
	for (i = 0; i < NumNodes; i++)
	{
		if (abs((long)X[i]) >= 0.99 || abs((long)Y[i]) >= 0.99)
			MyNode[i].Type = 1;
		else
			MyNode[i].Type = 0;
		MyNode[i].Jext = 0.0;
		MyNode[i].A = 0.0;
		MyNode[i].A1 = 0.0;
	}
	//mark boundary nodes and set Jext to 0 finished

	//Get Stiff and Dampinmg matrix and add to Node.Jext
	double x1, y1, x2, y2, x3, y3, Area;
	double b1, c1, b2, c2, b3, c3;
	double sigma, Jext;
	int I, J, K;
	double G01, G20, G12;
	for (i = 0; i < NumElems; i++)
	{
		sigma = 1.0;
		MyElem[i].Ve = 1.0;
		Jext = 0.0;
		if (MyElem[i].Type != -1000)
		{
			sigma = 1.0;
			MyElem[i].Ve = 1.0;
			Jext = 3.21;
		}
		I = MyElem[i].I; J = MyElem[i].J; K = MyElem[i].K;
		x1 = X[I]; x2 = X[J]; x3 = X[K];
		y1 = Y[I]; y2 = Y[J]; y3 = Y[K];
		Area = 0.5*(x1*(y2 - y3) + x2 * (y3 - y1) + x3 * (y1 - y2));
		MyElem[i].Area = Area;
		b1 = y2 - y3; c1 = x3 - x2;
		b2 = y3 - y1; c2 = x1 - x3;
		b3 = y1 - y2; c3 = x2 - x1;

		MyElem[i].Dp1 = (float)(sigma *Area / 12.0 / Delt);
		MyElem[i].Dp2 = (float)(sigma * Area / 6.0 / Delt);

		for (int ii = 0; ii < 3; ii++)
			for (int jj = 0; jj < 3; jj++)
				MyElem[i].Dp[ii][jj] = MyElem[i].Dp1;
		for (int ii = 0; ii < 3; ii++)
			MyElem[i].Dp[ii][ii] = MyElem[i].Dp2;

		G01 = 1.0 / 4  / Area * (b1 * b2 + c1 * c2);
		G20 = 1.0 / 4 / Area * (b1 * b3 + c1 * c3);
		G12 = 1.0 / 4 / Area * (b3 * b2 + c3 * c2);

		MyElem[i].Kc[0][0] = (float)(-G01 - G20);
		MyElem[i].Kc[0][1] = (float)G01;
		MyElem[i].Kc[0][2] = (float)G20;

		MyElem[i].Kc[1][0] = (float)G01;
		MyElem[i].Kc[1][1] = (float)(-G01 - G12);
		MyElem[i].Kc[1][2] = (float)G12;

		MyElem[i].Kc[2][0] = (float)G20;
		MyElem[i].Kc[2][1] = (float)G12;
		MyElem[i].Kc[2][2] = (float)(-G20 - G12);

		MyNode[I].Jext += (float)Area / 3.0*Jext;
		MyNode[J].Jext += (float)Area / 3.0*Jext;
		MyNode[K].Jext += (float)Area / 3.0*Jext;

	}

	for (i = 0; i < NumNodes; i++)
		if (MyNode[i].Type == 1)
			MyNode[i].Jext = 0.0;
	//Get Stiff and Dampinmg matrix and add to Node.Jext done

	free(X); free(Y);
}
void GPUMallocCopy()
{
	hipMalloc((void**)&d_WannaUpdateVe, sizeof(int));
	hipMalloc((void**)&d_NumElems, sizeof(int));
	hipMalloc((void**)&d_NumNodes, sizeof(int));
	hipMalloc((void**)&d_MyElem, NumElems * sizeof(FEMElem));
	hipMalloc((void**)&d_MyNode, NumNodes * sizeof(FEMNode));
	hipMemcpy(d_MyElem, MyElem, NumElems * sizeof(FEMElem), hipMemcpyHostToDevice);
	hipMemcpy(d_MyNode, MyNode, NumNodes * sizeof(FEMNode), hipMemcpyHostToDevice);
	hipMemcpy(d_NumNodes, &NumNodes, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_NumElems, &NumElems, sizeof(int), hipMemcpyHostToDevice);


	//message vecs between FEM and PCG
	hipMalloc((void**)&d_p, NumNodes * sizeof(float));

	hipMalloc((void**)&d_x, NumNodes * sizeof(float));
	float *hhh = (float *)malloc(NumNodes * sizeof(float));
	for (int ii = 0; ii < NumNodes; ii++)
		hhh[ii] = 0.0;
	hipMemcpy(d_x, hhh, NumNodes * sizeof(float), hipMemcpyHostToDevice);
	free(hhh);

	//message vecs between FEM and PCG

	//PCG vars
	//-------single  vars
	hipMalloc((void**)&d_alpha, sizeof(float));
	hipMalloc((void**)&d_beta, sizeof(float));
	hipMalloc((void**)&d_pap, sizeof(float));
	hipMalloc((void**)&d_rz1, sizeof(float));
	hipMalloc((void**)&d_rz0, sizeof(float));
	//-------reduction buffers
	hipMalloc((void**)&d_buffer1, NumNodes * sizeof(float));
	hipMalloc((void**)&d_buffer2, NumNodes * sizeof(float));
	//-------vector var package
	hipMalloc((void**)&d_PcgVec, NumNodes * sizeof(PcgVec));
	PcgVec *aaa = (PcgVec *)malloc(NumNodes * sizeof(PcgVec));
	for (int ii = 0; ii < NumNodes; ii++)
		aaa[ii].PreCondi = 1.0;
	hipMemcpy(d_PcgVec, aaa, NumNodes * sizeof(PcgVec), hipMemcpyHostToDevice);
	free(aaa);

	hipMalloc((void**)&d_yinuonuo, 6000 * sizeof(float));
	hipMalloc((void**)&d_anuonuo, NumNodes * sizeof(float));
}
void GPUFree()
{
	hipFree(d_MyElem);
	hipFree(d_MyNode);
	hipFree(d_NumNodes);
	hipFree(d_NumElems);
	hipFree(d_WannaUpdateVe);
	hipFree(d_x);
	hipFree(d_p);

	//PCG vars
	hipFree(d_x);
	hipFree(d_p);
	hipFree(d_buffer1);
	hipFree(d_buffer2);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(d_pap);
	hipFree(d_rz1);
	hipFree(d_rz0);
	hipFree(d_PcgVec);
	hipFree(d_yinuonuo);
	hipFree(d_anuonuo);

}



//NDD matrix multi Funcs Vec A*x or A*p
__global__ void ElmentLocal_MtxA_Dot_Vector(FEMElem* d_MyElem, FEMNode* d_MyNode, int *d_NumElems, float *Vector)
{

	int e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElems)
		return;

	int i, j;
	float ALocal[3];
	ALocal[0] = Vector[d_MyElem[e].I];
	ALocal[1] = Vector[d_MyElem[e].J];
	ALocal[2] = Vector[d_MyElem[e].K];
	for (i = 0; i < 3; i++)
	{
		d_MyElem[e].MatrixMultiVecBuff[i] = 0;
		for (j = 0; j < 3; j++)
			d_MyElem[e].MatrixMultiVecBuff[i] += (d_MyElem[e].Ve*d_MyElem[e].Kc[i][j] + d_MyElem[e].Dp[i][j]) * ALocal[j];
	}
}
__global__ void EachNodeExtractFromElemBuffInto_Ap(FEMElem* d_MyElem, FEMNode* d_MyNode, PcgVec *d_PcgVec, float *Vector, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	int e, LocalPos, i;
	float temp = 0;
	if (d_MyNode[nd].Type == 0)
	{
		temp = 0.0;
		for (i = 0; i < d_MyNode[nd].NumEle; i++)
		{
			e = d_MyNode[nd].EleID[i];
			LocalPos = d_MyNode[nd].EleOrd[i];
			temp += d_MyElem[e].MatrixMultiVecBuff[LocalPos];
		}
	}
	else
		temp = Vector[nd];
	d_PcgVec[nd].Ap = temp;

}

//math opertations to Vector and Number
__global__ void Num3isNum1dividedbyNum2(float *Num3, float *Num1, float *Num2)
{
	*Num3 = (*Num1) / (*Num2);
}
//-----------aim=sum_Vector
__global__ void reduction(float * Vector, float* aim, int *d_NumNodes) {
	__shared__ float sdata[5120];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;

	for (int ii = 0; ii < 40; ii++)
		sdata[tid + 128 * ii] = 0.0;

	for (int ii = 0; ii < 40; ii++)
		if ((tid + 128 * ii) < (*d_NumNodes))
			sdata[tid + 128 * ii] = Vector[tid + 128 * ii];
	__syncthreads();

	for (int ii = 1; ii < 40; ii++)
		sdata[tid] += sdata[tid + 128 * ii];
	__syncthreads();
	if (tid < 32)
	{
		for (int ii = 1; ii < 4; ii++)
			sdata[tid] += sdata[tid + 32 * ii];
	}

	__syncthreads();
	if (tid == 0)
	{
		for (int ii = 1; ii < 32; ii++)
			sdata[0] += sdata[ii];
		*aim = sdata[0];
	}
}
__global__ void reduction_2X(float * Vector1, float* aim1, float * Vector2, float* aim2, int *d_NumNodes) {
	 __shared__ float sdata1[5120];
	 __shared__ float sdata2[5120];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;

	for (int ii = 0; ii < 40; ii++)
	{
		sdata1[tid + 128 * ii] = 0.0;
		sdata2[tid + 128 * ii] = 0.0;
	}

	for (int ii = 0; ii < 40; ii++)
		if ((tid + 128 * ii) < (*d_NumNodes))
		{
			sdata1[tid + 128 * ii] = Vector1[tid + 128 * ii];
			sdata2[tid + 128 * ii] = Vector2[tid + 128 * ii];
		}
	__syncthreads();
	
	for (int ii = 1; ii < 40; ii++)
	{
		sdata1[tid] += sdata1[tid + 128 * ii];
		sdata2[tid] += sdata2[tid + 128 * ii];
	}
	__syncthreads();
	if (tid < 32)
	{
		for (int ii = 1; ii < 4; ii++)
		{
			sdata1[tid] += sdata1[tid + 32 * ii];
			sdata2[tid] += sdata2[tid + 32 * ii];
		}
	}

	__syncthreads();
	if (tid == 0)
	{
		for (int ii = 1; ii < 32; ii++)
		{
			sdata1[0] += sdata1[ii];
			sdata2[0] += sdata2[ii];
		}
		*aim1 = sdata1[0];
		*aim2 = sdata2[0];
	}
}


// FEM JExcitation->PcgVec.b
__global__ void UpdateExcitationSumTo_PCGSolver(FEMNode* d_MyNode, PcgVec *d_PcgVec, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;

	d_PcgVec[nd].b = d_MyNode[nd].Jext + d_MyNode[nd].TLM_Eq_Jsource + d_MyNode[nd].Jeddy;
}





float yinuonuo1[6000];
//PGC core Funcs
//----------make M=diag(A)^-1
__global__ void GPUmakePreconditoner(FEMElem* d_MyElem, FEMNode* d_MyNode, int *d_NumNodes, PcgVec *d_PcgVec)
{

	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	float temp;
	int e, LocalPos, i;
	if (d_MyNode[nd].Type == 0)
	{
		temp = 0.0;
		for (i = 0; i < d_MyNode[nd].NumEle; i++)
		{
			e = d_MyNode[nd].EleID[i];
			LocalPos = d_MyNode[nd].EleOrd[i];
			temp += (d_MyElem[e].Kc[LocalPos][LocalPos] * d_MyElem[e].Ve + d_MyElem[e].Dp[LocalPos][LocalPos]);
		}
		d_PcgVec[nd].PreCondi = 1.0 / temp;
	}

}
//----------initalize Pcg vectors based on x
__global__ void GPU_PCG_Prepare(int *d_NumNodes, float *d_p, PcgVec *d_PcgVec)
{

	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	float r0, z0;

	r0 = d_PcgVec[nd].b - d_PcgVec[nd].Ap;
	//r_old = (b - Ax);
	z0 = d_PcgVec[nd].PreCondi * r0;
	//z_old = M * r_old;
	d_PcgVec[nd].r0 = r0;
	//r_new = r_old;
	d_PcgVec[nd].r1 = r0;
	d_PcgVec[nd].z0 = z0;
	//z_new = z_old;
	d_PcgVec[nd].z1 = z0;
	//p = z_old;
	d_p[nd] = z0;

}
__global__ void Set0the_d_x(float *d_x, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	d_x[nd] = 0;
}

void CPU_PCG_Prepare()
{
	Set0the_d_x << <CudaBlckNum, CudaThrdNum >> > (d_x, d_NumNodes);
	GPUmakePreconditoner << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_NumNodes, d_PcgVec);
	//PcgVec.Ap=Ax;
	ElmentLocal_MtxA_Dot_Vector << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_NumElems, d_x);
	EachNodeExtractFromElemBuffInto_Ap << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_PcgVec, d_x, d_NumNodes);
	GPU_PCG_Prepare << <CudaBlckNum, CudaThrdNum >> > (d_NumNodes, d_p, d_PcgVec);
}

//----------PCG middle processes. seprated into differnet functions because need sync
__global__ void pAp_into_Buffer1_And_r0z0_into_Buffer2(PcgVec *d_PcgVec, int *d_NumNodes, float *d_p, float *d_buffer1, float *d_buffer2)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	d_buffer1[nd] = (d_PcgVec[nd].Ap)*d_p[nd];
	d_buffer2[nd] = (d_PcgVec[nd].r0)*(d_PcgVec[nd].z0);
}
__global__ void r1z1_into_Buffer1(PcgVec *d_PcgVec, int *d_NumNodes, float *d_buffer1)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	d_buffer1[nd] = (d_PcgVec[nd].r1)*(d_PcgVec[nd].z1);
}
__global__ void GPU_PCG_Process1(PcgVec *d_PcgVec, float *d_p, float* d_x, float *d_alpha, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	float r1;
	//x = x + alpha * p;
	d_x[nd] += (*d_alpha)*d_p[nd];
	//r_new = r_old - alpha * A * p;
	r1 = d_PcgVec[nd].r0 - (*d_alpha)* d_PcgVec[nd].Ap;
	d_PcgVec[nd].r1 = r1;
	//z_new = M * r_new;
	d_PcgVec[nd].z1 = d_PcgVec[nd].PreCondi * r1;
}
__global__ void GPU_PCG_Process2(PcgVec *d_PcgVec, float *d_p, float* d_x, float *d_beta, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	//p = z_new + beta *p;
	d_p[nd] = d_PcgVec[nd].z1 + (*d_beta)*d_p[nd];
	//r_old = r_new;
	d_PcgVec[nd].r0 = d_PcgVec[nd].r1;
	//z_old= z_new;
	d_PcgVec[nd].z0 = d_PcgVec[nd].z1;
}
//---------PCGsolver call
void PCGSolve_Ax_b()
{
	//alpha = r_old.' * z_old/(p.' * A * p);
	//-----------PcgVec.Ap=A*p;
	ElmentLocal_MtxA_Dot_Vector << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_NumElems, d_p);
	EachNodeExtractFromElemBuffInto_Ap << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_PcgVec, d_p, d_NumNodes);
	//-----------buffer1=pAp buffer2=r0dotz0
	pAp_into_Buffer1_And_r0z0_into_Buffer2 << <CudaBlckNum, CudaThrdNum >> > (d_PcgVec, d_NumNodes, d_p, d_buffer1, d_buffer2);
	//-----------pap=sum buffer1 rz0==sum buffer2
	reduction_2X << <1, 128 >> > (d_buffer1, d_pap, d_buffer2, d_rz0, d_NumNodes);
	//-----------alpha=rz0/pap
	Num3isNum1dividedbyNum2 << <1, 1 >> > (d_alpha, d_rz0, d_pap);
	//hipMemcpy(&yinuonuo1, d_alpha, sizeof(float), hipMemcpyDeviceToHost);

	//x = x + alpha * p;
	//r_new = r_old - alpha * A * p;
	//z_new = M * r_new;
	GPU_PCG_Process1 << <CudaBlckNum, CudaThrdNum >> > (d_PcgVec, d_p, d_x, d_alpha, d_NumNodes);
	//hipMemcpy(&yinuonuo1, d_x, NumNodes * sizeof(float), hipMemcpyDeviceToHost);

	//beta = (z_new.'  * r_new)/(z_old.' * r_old);
	//-----------buffer1=z1*r1;
	r1z1_into_Buffer1 << <CudaBlckNum, CudaThrdNum >> > (d_PcgVec, d_NumNodes, d_buffer1);
	//-----------rz1=sum_buffer1
	reduction << <1, 128 >> > (d_buffer1, d_rz1, d_NumNodes);
	//-----------beta=rz1/rz0
	Num3isNum1dividedbyNum2 << <1, 1 >> > (d_beta, d_rz1, d_rz0);
	//hipMemcpy(&yinuonuo1, d_beta, sizeof(float), hipMemcpyDeviceToHost);

	//p = z_new + beta *p;
	//r_old = r_new;
	//z_old= z_new;
	GPU_PCG_Process2 << <CudaBlckNum, CudaThrdNum >> > (d_PcgVec, d_p, d_x, d_beta, d_NumNodes);
}

//TLM Funcs
__device__ void GaussElimSolve(float *X, float Mtrx[3][3], float *RHS)//Mtrx changed after use X=Mtrx\RHS;
{

	int i, j, k; float Aik; float S;
	for (k = 0; k < 2; k++)
	{
		if (!Mtrx[k][k])
			printf("Matrix is not good\n");
		for (i = k + 1; i < 3; i++)
		{
			Aik = Mtrx[i][k] / Mtrx[k][k];
			for (j = k; j < 3; j++)
			{
				Mtrx[i][j] = Mtrx[i][j] - Aik * Mtrx[k][j];
			}
			RHS[i] = RHS[i] - Aik * RHS[k];
		}
	}

	X[2] = RHS[2] / Mtrx[2][2];
	for (k = 1; k >= 0; k--)
	{
		S = RHS[k];
		for (j = k + 1; j < 3; j++)
		{
			S = S - Mtrx[k][j] * X[j];
		}
		X[k] = S / Mtrx[k][k];
	}



}
//-----------  Vi[3]---inject--->ElementReistors---reflect---->Vr[3] and return value: Ve at given Vi
__device__ float Newton_Raphson_Element(float *Vr, float *Vi, float Ve, float *Kc, float *B2terms)
//Vi is x0,y,z0 Vr is x y z // Kc+Ve restore G and Y //B2terms+X+Y restore B^2 Thus mag flux density B
{


	int count = 0, i, j;
	float dVr[3];
	float Y[3];
	float B = 0, B2;
	float dB2_dVr[3];
	float VeAtB, dVeAtB_dB2;

	float Jacobian[3][3];
	float Residual[3];

	if (Vi[0] == 0 && Vi[1] == 0 && Vi[2] == 0)
	{
		Vr[0] = 0; Vr[1] = 0; Vr[2] = 0;
		return Ve;
	}
	else
	{
		for (i = 0; i < 3; i++)
			Y[i] = Kc[i] * Ve;//ʵ���и����š�ֻ��Ϊ�˺�ԭ���ĳ��򱣳�һ��

		while (count < 10)
			//while(count<10)
		{
			//for (i = 0; i < 3; i++)
				//Vrtemp[i] = Vr[i];
			B2 = 0;
			for (i = 0; i < 3; i++)
				B2 += B2terms[i] * (Vi[i] + Vr[i])*(Vi[i] + Vr[i]);
			B = sqrt(B2);
			//---------------------------------------modify nonliear curve here below
			if (B < 0.6)
			{
				VeAtB = 2.0;
				dVeAtB_dB2 = 0.0;
			}
			else
			{
				VeAtB = 2.0 + 1e5 * (B - 0.6) * (B - 0.6)* (B - 0.6)/ B;
				dVeAtB_dB2 = (B * 3e5 * (B - 0.6) * (B - 0.6)- 1e5 * (B - 0.6)* (B - 0.6)* (B - 0.6)) / B / B / 2 / B;
			}
			//---------------------------------------modify nonliear curve here above

			//--------make residual and Jacobian matrix
			for (i = 0; i < 3; i++)
				Residual[i] = (VeAtB*Kc[i] * (Vi[i] + Vr[i]) - Y[i] * (Vi[i] - Vr[i]));

			for (i = 0; i < 3; i++)
				dB2_dVr[i] = (Vi[i] + Vr[i]) * 2 * B2terms[i];

			for (i = 0; i < 3; i++)
				for (j = 0; j < 3; j++)
					Jacobian[i][j] = -Kc[i] * (Vi[i] + Vr[i])*dVeAtB_dB2*dB2_dVr[j];
			for (i = 0; i < 3; i++)
				Jacobian[i][i] -= Kc[i] * VeAtB + Y[i];
			//--------solve for Vr step dVr=Jcb\Rsd
			GaussElimSolve(dVr, Jacobian, Residual);
			for (i = 0; i < 3; i++)
				Vr[i] += dVr[i];

			count++;
		}
	}

	//printf("%f %f\n", Vi[0], Vr[0]);
	return VeAtB;
}

__global__ void NonlinearElementsUpdate_Ve_And_EquvJSource(FEMElem* d_MyElem, int *d_NumElems, float *d_x, int *d_WannaUpdateVe)
{

	int e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElems)
		return;
	//	if (d_MyElem[e].Type != 1)//nonliaer elements geometry group
		//	return;
		//Local Node ID
	int I = d_MyElem[e].I; int J = d_MyElem[e].J; int K = d_MyElem[e].K;

	float Vr[3];//copy to Local
	Vr[0] = d_MyElem[e].Vr[0];
	Vr[1] = d_MyElem[e].Vr[1];
	Vr[2] = d_MyElem[e].Vr[2];

	//Collect Inject wave amplitude from the global network (volatge)
	float Vi[3];
	Vi[0] = (d_x[J] - d_x[I] - Vr[0]);
	Vi[1] = (d_x[K] - d_x[J] - Vr[1]);
	Vi[2] = (d_x[I] - d_x[K] - Vr[2]);
	//NR solve reflection waves
	float Kc[3];// 3 Kc terms to calculate G func and Y value in the 3*3 nonliear matrix group
	Kc[0] = d_MyElem[e].Kc[0][1];// ----------note minus sign!!!!!!!!!!!
	Kc[1] = d_MyElem[e].Kc[1][2];
	Kc[2] = d_MyElem[e].Kc[2][0];
	float B2terms[3];// ---------restore B^2 from Vr Vi
	B2terms[0] = -Kc[0] / d_MyElem[e].Area;
	B2terms[1] = -Kc[1] / d_MyElem[e].Area;
	B2terms[2] = -Kc[2] / d_MyElem[e].Area;
	float Ve = Newton_Raphson_Element(Vr, Vi, d_MyElem[e].Ve, Kc, B2terms);// ----------elemental solve
				//for (int ii = 0; ii < 3; ii++)//linear test replcement for above func
				//Vr[ii] = -Vi[ii] / 3.0;
	//Update Ve here if in need
	//chagne reflected volage waves into eqiuvalent inject current sources on 3 element nodes to ground
	float temp1, temp2, temp3;
	temp1 = Kc[0] * d_MyElem[e].Ve * Vr[0] * 2;
	temp2 = Kc[1] * d_MyElem[e].Ve * Vr[1] * 2;
	temp3 = Kc[2] * d_MyElem[e].Ve * Vr[2] * 2;
	d_MyElem[e].Ieq[0] = temp1 - temp3;
	d_MyElem[e].Ieq[1] = temp2 - temp1;
	d_MyElem[e].Ieq[2] = temp3 - temp2;
	d_MyElem[e].Vr[0] = Vr[0];
	d_MyElem[e].Vr[1] = Vr[1];
	d_MyElem[e].Vr[2] = Vr[2];
	if (*d_WannaUpdateVe == 1)
		d_MyElem[e].Ve = Ve;
}

__global__ void EachNodeExtractFromElemIeqInto_NodeIeq(FEMElem* d_MyElem, FEMNode* d_MyNode, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	int e, LocalPos, i;
	float temp = 0;
	if (d_MyNode[nd].Type == 0)
	{
		temp = 0.0;
		for (i = 0; i < d_MyNode[nd].NumEle; i++)
		{
			e = d_MyNode[nd].EleID[i];
			LocalPos = d_MyNode[nd].EleOrd[i];
			temp += d_MyElem[e].Ieq[LocalPos];
		}
	}
	else
		temp = 0.0;
	d_MyNode[nd].TLM_Eq_Jsource = temp;

}





PcgVec PcgV[320];

void TLM_MatrixFree_Solve_for_this_timepoint()
{
	int wannaupdateve = 0;
	hipMemcpy(d_WannaUpdateVe, &wannaupdateve, sizeof(int), hipMemcpyHostToDevice);
	for (int TLMct = 0; TLMct < 6; TLMct++)
	{
		if (TLMct == 5)
		{
			wannaupdateve = 0;
			hipMemcpy(d_WannaUpdateVe, &wannaupdateve, sizeof(int), hipMemcpyHostToDevice);
		}
		UpdateExcitationSumTo_PCGSolver << <CudaBlckNum, CudaThrdNum >> > (d_MyNode, d_PcgVec, d_NumNodes);
		hipMemcpy(&PcgV, d_PcgVec, 320 * sizeof(PcgVec), hipMemcpyDeviceToHost);

		CPU_PCG_Prepare();
		for (int pcgit = 0; pcgit < 40; pcgit++)
			PCGSolve_Ax_b();
		NonlinearElementsUpdate_Ve_And_EquvJSource << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_NumElems, d_x, d_WannaUpdateVe);
		EachNodeExtractFromElemIeqInto_NodeIeq << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_NumNodes);
		hipMemcpy(&MyNode, d_MyNode, NumNodes * sizeof(FEMNode), hipMemcpyDeviceToHost);
		hipMemcpy(&yinuonuo1, d_x, NumNodes * sizeof(float), hipMemcpyDeviceToHost);
		//printf("%f\n", yinuonuo1[2]);
		//		for (int i = 0; i < 320; i++)
		//		printf("  %f", MyNode[i].TLM_Eq_Jsource);
	}
}


//Jeddy=Dp*A;
__global__ void ElmentLocal_MtxDp_Dot_d_x(FEMElem* d_MyElem, FEMNode* d_MyNode, int *d_NumElems, float *d_x)
{

	int e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElems)
		return;

	int i, j;
	float ALocal[3];
	ALocal[0] = d_x[d_MyElem[e].I];
	ALocal[1] = d_x[d_MyElem[e].J];
	ALocal[2] = d_x[d_MyElem[e].K];
	for (i = 0; i < 3; i++)
	{
		d_MyElem[e].MatrixMultiVecBuff[i] = 0;
		for (j = 0; j < 3; j++)
			d_MyElem[e].MatrixMultiVecBuff[i] += (d_MyElem[e].Dp[i][j]) * ALocal[j];
	}
}
__global__ void EachNodeExtractFromElemBuffInto_Jeddy(FEMElem* d_MyElem, FEMNode* d_MyNode, float *d_x, int *d_NumNodes)
{
	int nd = threadIdx.x + blockIdx.x*blockDim.x;
	if (nd >= *d_NumNodes)
		return;
	int e, LocalPos, i;
	float temp = 0;
	if (d_MyNode[nd].Type == 0)
	{
		temp = 0.0;
		for (i = 0; i < d_MyNode[nd].NumEle; i++)
		{
			e = d_MyNode[nd].EleID[i];
			LocalPos = d_MyNode[nd].EleOrd[i];
			temp += d_MyElem[e].MatrixMultiVecBuff[LocalPos];
		}
	}
	else
		temp = d_x[nd];
	d_MyNode[nd].Jeddy = temp;

}



int main()
{

	LoadMeshInfoAndPrepareFEM();
	GPUMallocCopy();

	hipEvent_t start, stop;//unit: ms

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int t = 0; t < 10; t++)
	{
		ElmentLocal_MtxDp_Dot_d_x << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_NumElems, d_x);
		EachNodeExtractFromElemBuffInto_Jeddy << <CudaBlckNum, CudaThrdNum >> > (d_MyElem, d_MyNode, d_x, d_NumNodes);

		TLM_MatrixFree_Solve_for_this_timepoint();
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	


	hipMemcpy(&yinuonuo1, d_x, NumNodes * sizeof(float), hipMemcpyDeviceToHost);
	// for (int i = 0; i < 320; i++)
	// printf("%f \n", yinuonuo1[i]);


	printf("\nElapsed time for NDDR Iteration is %.4f ms\n", elapsedTime);



	GPUFree();



}