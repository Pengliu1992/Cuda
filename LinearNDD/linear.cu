#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#define Mu0 1.256637e-6
#define pi 3.1415927
#define MuFeCore 0.0020 //linear region of BH curve
#define MaxNode 8000
#define MaxElem 16000
#define Init0 0.0
#define NRCount 1

#define CudaThrdNum 128
#define CudaBlckNum 128

#ifndef _TIMER_H_
#define _TIMER_H_
typedef struct
{
	double Y;
	double Y1;
	double Y2;
}interp_t;
#define GET_TIME(now){ \
	double t; \
	t=clock(); \
	now =t/CLOCKS_PER_SEC ;\
}
#endif

typedef struct
{
	int Id;
	double x;
	double y;
	int NumEle;
	int EleID[10];
	int EleOrd[10];
	int NeiborNode[20];
	int NumNeiborNodes;
	double TotalArea;
	int Type;
	double A0;
	double A1;
	double K;
	double SumRHSContri; // Sum from all elems involved with this node
	double JsSum;
	double SumNeiborJsSum;//from Js
}FEMNode;
typedef struct
{
	int Id;
	int Nodes[3];
	double Area;
	int Type;
	double Ve;
	double Js;
	double sigma;
	double Me[3][3];
	double ElmRowSum[3][3];// weighted row sum by gamma1 for 3 nodes
	double RHSContri[3]; // RHS or b calculated by elmK*A on 3 nodes
}FEMElem;


//Host parameters for FEM
int NumNodes, NumElem;
FEMNode MyNode[MaxNode];
FEMElem MyElem[MaxElem];
double gamma1 = 100.0;
double CurrentDensity = 1e6;
//Device parameters for FEM
int *d_NumNodes, *d_NumElem;
double *d_gamma1;
double *d_CurrentDensity;
FEMNode *d_MyNode;
FEMElem *d_MyElem;

void LoadMeshInfo()
{
	FILE* ip;
	int i, flag = 0;
	char filename[50];

	char line[50];


	sprintf(filename, "3733.mphtxt");

	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}
	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# number of mesh points") != NULL)
		{
			sscanf(line, "%d", &(NumNodes));
		}
	}
	fclose(ip);


	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}

	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# Mesh point coordinates") != NULL)
		{
			for (i = 0; i < NumNodes; i++)
			{
				fgets(line, sizeof(line), ip);
				sscanf(line, "%lf %lf\n", &(MyNode[i].x), &(MyNode[i].y));
			}
		}
	}
	fclose(ip);



	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}

	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# number of elements") != NULL)
		{
			flag = flag + 1;
			if (flag == 3)
			{
				sscanf(line, "%d", &(NumElem));
				fgets(line, sizeof(line), ip);

				for (i = 0; i < NumElem; i++)
				{
					fgets(line, sizeof(line), ip);
					sscanf(line, "%d %d %d\n", &(MyElem[i].Nodes[0]), &(MyElem[i].Nodes[1]), &(MyElem[i].Nodes[2]));

				}
			}
		}
	}
	fclose(ip);

	flag = 0;
	if ((ip = fopen(filename, "r")) == NULL) {
		printf("error opening the input data.\n");
	}
	while (fgets(line, sizeof(line), ip) != NULL) {

		if (strstr(line, "# Geometric entity indices") != NULL)
		{
			flag = flag + 1;
			if (flag == 3)
			{
				for (i = 0; i < NumElem; i++)
				{
					fgets(line, sizeof(line), ip);
					sscanf(line, "%d\n", &(MyElem[i].Type));
					//	printf("\nType %d", MyElem[i].Type);
				}
			}
		}
	}
	fclose(ip);



}

void FEM_Host_Data_Prepare()
{
	int i, j, k = 0;
	double x1, x2, x3, y1, y2, y3;
	double b1, b2, b3, c1, c2, c3;
	static int dummy[MaxNode];
	LoadMeshInfo();

	//Boundary node ID and values
	for (i = 0; i < NumNodes; i++)
	{
		MyNode[i].A0 = Init0;
		MyNode[i].Type = 0;
		//MyNode[i].A0 = (double)i + 1.0;
		if (fabs(MyNode[i].x - 3.5) < 1e-5 || fabs(MyNode[i].x + 3.5) < 1e-5 || fabs(MyNode[i].y - 2.5) < 1e-5 || fabs(MyNode[i].y + 2.5) < 1e-5)

		{
			MyNode[i].Type = 1;


		}

	}

	//set initial Ve config
	for (i = 0; i < NumElem; i++)

	{
		MyElem[i].Js = 0;
		if (1)//MyElem[i].Type != 2)
			MyElem[i].Ve = 1.0 / Mu0;
		else
			MyElem[i].Ve = 1.0 / MuFeCore;

	}


	//Element K matrix 
	for (i = 0; i < NumElem; i++)
	{
		x1 = MyNode[MyElem[i].Nodes[0]].x; x2 = MyNode[MyElem[i].Nodes[1]].x; x3 = MyNode[MyElem[i].Nodes[2]].x;
		y1 = MyNode[MyElem[i].Nodes[0]].y; y2 = MyNode[MyElem[i].Nodes[1]].y; y3 = MyNode[MyElem[i].Nodes[2]].y;
		MyElem[i].Area = 0.5*(x1*(y2 - y3) + x2 * (y3 - y1) + x3 * (y1 - y2));
		b1 = y2 - y3; c1 = x3 - x2;
		b2 = y3 - y1; c2 = x1 - x3;
		b3 = y1 - y2; c3 = x2 - x1;

		MyElem[i].Me[0][0] = 1.0 / 4 / MyElem[i].Area*(b1*b1 + c1 * c1);
		MyElem[i].Me[0][1] = 1.0 / 4 / MyElem[i].Area*(b1*b2 + c1 * c2);
		MyElem[i].Me[0][2] = 1.0 / 4 / MyElem[i].Area*(b1*b3 + c1 * c3);

		MyElem[i].Me[1][0] = 1.0 / 4 / MyElem[i].Area*(b1*b2 + c1 * c2);
		MyElem[i].Me[1][1] = 1.0 / 4 / MyElem[i].Area*(b2*b2 + c2 * c2);
		MyElem[i].Me[1][2] = 1.0 / 4 / MyElem[i].Area*(b2*b3 + c2 * c3);

		MyElem[i].Me[2][0] = 1.0 / 4 / MyElem[i].Area*(b1*b3 + c1 * c3);
		MyElem[i].Me[2][1] = 1.0 / 4 / MyElem[i].Area*(b3*b2 + c3 * c2);
		MyElem[i].Me[2][2] = 1.0 / 4 / MyElem[i].Area*(b3*b3 + c3 * c3);

		//get ElmRowSum[i][...] is gamma1 weighted row sum for ith node. 
				//set 0
		for (j = 0; j < 3; j++)
			for (k = 0; k < 3; k++)
				MyElem[i].ElmRowSum[j][k] = 0;

		double temp;
		for (j = 0; j < 3; j++)
			for (k = 0; k < 3; k++)
				if (MyNode[MyElem[i].Nodes[k]].Type != 1) //first type bdry node row is set to 0
				{
					if (k == j)
						temp = 1.0;
					else
						temp = 1.0 / gamma1;
					MyElem[i].ElmRowSum[j][0] += temp * MyElem[i].Me[k][0];
					MyElem[i].ElmRowSum[j][1] += temp * MyElem[i].Me[k][1];
					MyElem[i].ElmRowSum[j][2] += temp * MyElem[i].Me[k][2];
				}

	}


	// topology link info of Nodes
	for (i = 0; i < NumNodes; i++)
	{
		MyNode[i].NumEle = 0;
		MyNode[i].TotalArea = 0;
		MyNode[i].NumNeiborNodes = 0;
	}
	for (i = 0; i < NumElem; i++)
		for (j = 0; j < 3; j++)
		{
			MyNode[MyElem[i].Nodes[j]].EleID[MyNode[MyElem[i].Nodes[j]].NumEle] = i;
			MyNode[MyElem[i].Nodes[j]].EleOrd[MyNode[MyElem[i].Nodes[j]].NumEle] = j;
			MyNode[MyElem[i].Nodes[j]].NumEle++;
			MyNode[MyElem[i].Nodes[j]].TotalArea = MyNode[MyElem[i].Nodes[j]].TotalArea + MyElem[i].Area;
		}
	for (i = 0; i < NumNodes; i++)
	{
		for (j = 0; j < NumNodes; j++)
			dummy[j] = 0;
		for (j = 0; j < MyNode[i].NumEle; j++)
			for (k = 0; k < 3; k++)
				dummy[MyElem[MyNode[i].EleID[j]].Nodes[k]] = 1;
		for (j = 0; j < NumNodes; j++)
			if (dummy[j] == 1 && j != i)
			{
				MyNode[i].NeiborNode[MyNode[i].NumNeiborNodes] = j;
				(MyNode[i].NumNeiborNodes)++;
			}

	}

	for (i = 0; i < NumNodes; i++)
	{
		if (MyNode[i].Type == 0)
		{
			MyNode[i].K = 0;
			for (j = 0; j < MyNode[i].NumEle; j++)
				MyNode[i].K = MyNode[i].K + MyElem[MyNode[i].EleID[j]].Me[MyNode[i].EleOrd[j]][MyNode[i].EleOrd[j]];
		}
	}
	printf("Number of Nodes: %d   Number of Elements: %d\n", NumNodes, NumElem);

}
__global__ void ApplyCurrent(double *d_CurrentDensity, int *d_NumElem, FEMElem* d_MyElem)
{
	int e;
	e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElem)
		return;
	if (d_MyElem[e].Type == 5)  d_MyElem[e].Js = (*d_CurrentDensity);
	if (d_MyElem[e].Type == 7)  d_MyElem[e].Js = -(*d_CurrentDensity);
}
__global__ void JsSumCalculate(int *d_NumNodes, FEMElem* d_MyElem, FEMNode* d_MyNode)
{
	int node = threadIdx.x + blockIdx.x*blockDim.x;
	if (node >= *d_NumNodes)
		return;
	int e;
	d_MyNode[node].JsSum = 0;
	for (e = 0; e < d_MyNode[node].NumEle; e++)
		d_MyNode[node].JsSum += (d_MyElem[d_MyNode[node].EleID[e]].Js)*(d_MyElem[d_MyNode[node].EleID[e]].Area) / 3;
}
__global__ void SumNeiborJsSumCalculate(int *d_NumNodes, FEMElem* d_MyElem, FEMNode* d_MyNode)
{
	int node = threadIdx.x + blockIdx.x*blockDim.x;
	if (node >= *d_NumNodes)
		return;
	int n;
	d_MyNode[node].SumNeiborJsSum = 0;
	for (n = 0; n < d_MyNode[node].NumNeiborNodes; n++)
		d_MyNode[node].SumNeiborJsSum += (d_MyNode[d_MyNode[node].NeiborNode[n]].JsSum);
}
__global__ void SumNodeRHSContriCalculate(int *d_NumNodes, FEMElem* d_MyElem, FEMNode* d_MyNode)
{
	int node = threadIdx.x + blockIdx.x*blockDim.x;
	if (node >= *d_NumNodes)
		return;
	int e;
	d_MyNode[node].SumRHSContri = 0;
	for (e = 0; e < d_MyNode[node].NumEle; e++)
		d_MyNode[node].SumRHSContri += d_MyElem[d_MyNode[node].EleID[e]].RHSContri[d_MyNode[node].EleOrd[e]];

}
__global__ void ElmRHSContriCalculate(int *d_NumElem, FEMElem* d_MyElem, FEMNode* d_MyNode)
{

	int e;
	e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElem)
		return;

	int row, col;
	for (row = 0; row < 3; row++)
		d_MyElem[e].RHSContri[row] = 0;
	for (row = 0; row < 3; row++)
	{
		for (col = 0; col < 3; col++)
			d_MyElem[e].RHSContri[row] += d_MyElem[e].Me[row][col] * d_MyNode[d_MyElem[e].Nodes[col]].A0;
		d_MyElem[e].RHSContri[row] = d_MyElem[e].RHSContri[row] * d_MyElem[e].Ve;
	}

}
__global__ void UpdateSolutiontoA1(int *d_NumNodes, FEMElem* d_MyElem, FEMNode* d_MyNode, double *d_gamma1)
{
	int node = threadIdx.x + blockIdx.x*blockDim.x;
	if (node >= *d_NumNodes)
		return;

	double RHS, LHS, dF_dA, NRsolition, temp;
	int i, j, NeiborID, e, LocalPos;
	double ALocal[3], B2, B, V, VB2, B2A;
	int NRct;
	if (d_MyNode[node].Type != 1)
	{
		//----------Get RHS
		RHS = 0;
		// subdomain RHS sum  = SumofAllneibornodes-Sumof subdomain elems's contribution on neibornodes 
		for (i = 0; i < d_MyNode[node].NumNeiborNodes; i++)
		{
			NeiborID = d_MyNode[node].NeiborNode[i];
			if (d_MyNode[NeiborID].Type != 1)
				RHS -= d_MyNode[NeiborID].SumRHSContri;
		}

		for (i = 0; i < d_MyNode[node].NumEle; i++)
		{
			e = d_MyNode[node].EleID[i];
			for (j = 0; j < 3; j++)
				if (node != d_MyElem[e].Nodes[j] && d_MyNode[d_MyElem[e].Nodes[j]].Type != 1)
					RHS += d_MyElem[e].RHSContri[j];
		}

		RHS = (RHS + d_MyNode[node].SumNeiborJsSum) / (*d_gamma1) + d_MyNode[node].JsSum;
		//----------Get LHS and dF_dA

		NRsolition = d_MyNode[node].A0;
		//----------NRiteration
		for (NRct = 0; NRct < NRCount; NRct++)
		{
			LHS = 0;
			dF_dA = 0;
			for (i = 0; i < d_MyNode[node].NumEle; i++)
			{
				e = d_MyNode[node].EleID[i];
				LocalPos = d_MyNode[node].EleOrd[i];
				ALocal[0] = d_MyNode[d_MyElem[e].Nodes[0]].A0;
				ALocal[1] = d_MyNode[d_MyElem[e].Nodes[1]].A0;
				ALocal[2] = d_MyNode[d_MyElem[e].Nodes[2]].A0;
				ALocal[LocalPos] = NRsolition;
				if (0)
				{
					B2 = -1 / d_MyElem[e].Area*(d_MyElem[e].Me[0][1] * pow(ALocal[0] - ALocal[1], 2)
						+ d_MyElem[e].Me[1][2] * pow(ALocal[1] - ALocal[2], 2)
						+ d_MyElem[e].Me[2][0] * pow(ALocal[2] - ALocal[0], 2));
					B = sqrt(B2);

					if (B <= 0.6)
					{
						for (j = 0; j < 3; j++)
							LHS += ALocal[j] * d_MyElem[e].ElmRowSum[LocalPos][j] / MuFeCore;
						dF_dA += d_MyElem[e].ElmRowSum[LocalPos][LocalPos] / MuFeCore;
						//		if (NRct == (NRCount - 1))
								//	d_MyElem[e].Ve = 1 / MuFeCore;
					}
					else
					{
						V = 1 / MuFeCore + 3000.0*pow(B - 0.6, 3) / B;
						VB2 = (B*9000.0*pow(B - 0.6, 2) - 3000.0*pow(B - 0.6, 3)) / B / B / 2 / B;
						B2A = 0;

						for (j = 0; j < 3; j++)
							if (j != LocalPos)
								B2A = B2A + d_MyElem[e].Me[LocalPos][j] * (ALocal[LocalPos] - ALocal[j]);
						B2A = -B2A * 2 / d_MyElem[e].Area;
						// LHS+=V* Alocal dot ElmRowSum(Localpos,:) 
						temp = 0;
						for (j = 0; j < 3; j++)
						{
							dF_dA += VB2 * B2A * ALocal[j] * d_MyElem[e].ElmRowSum[LocalPos][j];
							temp += ALocal[j] * d_MyElem[e].ElmRowSum[LocalPos][j];
						}
						dF_dA += V * d_MyElem[e].ElmRowSum[LocalPos][LocalPos];
						LHS += temp * V;
						//		if (NRct == (NRCount - 1))
							//		d_MyElem[e].Ve = V;
					}
				}
				else
				{
					for (j = 0; j < 3; j++)
						LHS += ALocal[j] * d_MyElem[e].ElmRowSum[LocalPos][j] / Mu0;
					dF_dA += d_MyElem[e].ElmRowSum[LocalPos][LocalPos] / Mu0;
				}
			}
			NRsolition += (RHS - LHS) / dF_dA;
			//	printf("\nB%e NR%e RHS%e LHS%e dFda %e\n",B, NRsolition, RHS,LHS, dF_dA);
		}
		d_MyNode[node].A1 = NRsolition;

	}
}
__global__ void CopyA1ToA0(int *d_NumNodes, FEMElem* d_MyElem, FEMNode* d_MyNode)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= *d_NumNodes)
		return;
	d_MyNode[i].A0 = d_MyNode[i].A1;
}
__global__ void UpdateVe(int *d_NumElem, FEMElem* d_MyElem, FEMNode* d_MyNode)
{
	int e;
	e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElem || 1)
		return;
	double ALocal[3], B2, B, V;
	ALocal[0] = d_MyNode[d_MyElem[e].Nodes[0]].A0;
	ALocal[1] = d_MyNode[d_MyElem[e].Nodes[1]].A0;
	ALocal[2] = d_MyNode[d_MyElem[e].Nodes[2]].A0;
	B2 = -1 / d_MyElem[e].Area*(d_MyElem[e].Me[0][1] * pow(ALocal[0] - ALocal[1], 2)
		+ d_MyElem[e].Me[1][2] * pow(ALocal[1] - ALocal[2], 2)
		+ d_MyElem[e].Me[2][0] * pow(ALocal[2] - ALocal[0], 2));
	B = sqrt(B2);
	if (B <= 0.6)
		V = 1 / MuFeCore;
	else
		V = 1 / MuFeCore + 3000.0*pow(B - 0.6, 3) / B;
	d_MyElem[e].Ve = V;
}
__global__ void Updategamma1OnDevice(int *d_NumElem, FEMElem* d_MyElem, FEMNode* d_MyNode, double *d_gamma1)
{
	int e;
	e = threadIdx.x + blockIdx.x*blockDim.x;
	if (e >= *d_NumElem)
		return;
	int j, k;
	for (j = 0; j < 3; j++)
		for (k = 0; k < 3; k++)
			d_MyElem[e].ElmRowSum[j][k] = 0;

	double temp;
	for (j = 0; j < 3; j++)
		for (k = 0; k < 3; k++)
			if (d_MyNode[d_MyElem[e].Nodes[k]].Type != 1) //first type bdry node row is set to 0
			{
				if (k == j)
					temp = 1.0;
				else
					temp = 1.0 / (*d_gamma1);
				d_MyElem[e].ElmRowSum[j][0] += temp * d_MyElem[e].Me[k][0];
				d_MyElem[e].ElmRowSum[j][1] += temp * d_MyElem[e].Me[k][1];
				d_MyElem[e].ElmRowSum[j][2] += temp * d_MyElem[e].Me[k][2];
			}
}

void GPUInitialMallocCopy()
{
	//cuda initiallze
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
		int max_multiprocessors = 0;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				
			}
		}

	}
	hipSetDevice(0);
	hipMalloc((void**)&d_MyNode, NumNodes * sizeof(FEMNode));
	hipMalloc((void**)&d_MyElem, NumElem * sizeof(FEMElem));
	hipMalloc((void**)&d_NumNodes, sizeof(int));
	hipMalloc((void**)&d_NumElem, sizeof(int));
	hipMalloc((void**)&d_gamma1, sizeof(double));
	hipMalloc((void**)&d_CurrentDensity, sizeof(double));
	hipMemcpy(d_MyNode, MyNode, NumNodes * sizeof(FEMNode), hipMemcpyHostToDevice);
	hipMemcpy(d_MyElem, MyElem, NumElem * sizeof(FEMElem), hipMemcpyHostToDevice);
	hipMemcpy(d_NumNodes, &NumNodes, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_NumElem, &NumElem, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_gamma1, &gamma1, sizeof(double), hipMemcpyHostToDevice);
}
void GPUFree()
{
	hipFree(d_NumElem);
	hipFree(d_NumNodes);
	hipFree(d_MyElem);
	hipFree(d_MyNode);
	hipFree(d_gamma1);
	hipFree(d_CurrentDensity);
}
//for the test
double My_abs(double x)
{
	if (x >= 0)
		return x;
	else
		return -x;
}
double PorbeValueVsItration[4000];
void MyUpdategamma1(double danteng)
{
	hipMemcpy(d_gamma1, &danteng, sizeof(double), hipMemcpyHostToDevice);
	Updategamma1OnDevice << <CudaBlckNum, CudaThrdNum >> > (d_NumElem, d_MyElem, d_MyNode, d_gamma1);
}
int main() {

	FEM_Host_Data_Prepare();
	GPUInitialMallocCopy();
	int probeID = -1;
	//find the beloved probe node
	for (int i = 1; i < NumNodes; i++)
		if (My_abs(0.25 - MyNode[i].x) < 1e-13)
			if (My_abs(1.30 - MyNode[i].y) < 1e-13)
				probeID = i;
	printf("\nprobeID=%d\n", probeID);


	//Set currentdensity on device
	CurrentDensity = 1e6;
	hipMemcpy(d_CurrentDensity, &CurrentDensity, sizeof(double), hipMemcpyHostToDevice);
	ApplyCurrent << <CudaBlckNum, CudaThrdNum >> > (d_CurrentDensity, d_NumElem, d_MyElem);

	//Js contribution prepare
	JsSumCalculate << <CudaBlckNum, CudaThrdNum >> > (d_NumNodes, d_MyElem, d_MyNode);
	SumNeiborJsSumCalculate << <CudaBlckNum, CudaThrdNum >> > (d_NumNodes, d_MyElem, d_MyNode);
	//gamma1 prepare
	gamma1 = 5.0;
	hipMemcpy(d_gamma1, &gamma1, sizeof(double), hipMemcpyHostToDevice);
	Updategamma1OnDevice << <CudaBlckNum, CudaThrdNum >> > (d_NumElem, d_MyElem, d_MyNode, d_gamma1);

	hipEvent_t start, stop;//unit: ms

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (int RelaxCount = 0; RelaxCount < 1000; RelaxCount++)
	{
		//if (RelaxCount == 2000) MyUpdategamma1(4.9);
		//if (RelaxCount ==40) MyUpdategamma1(3.3);
		//if (RelaxCount == 30) MyUpdategamma1(5.0);
		//if (RelaxCount == 2000) MyUpdategamma1(6.2);
		ElmRHSContriCalculate << <CudaBlckNum, CudaThrdNum >> > (d_NumElem, d_MyElem, d_MyNode);

		SumNodeRHSContriCalculate << <CudaBlckNum, CudaThrdNum >> > (d_NumNodes, d_MyElem, d_MyNode);

		UpdateSolutiontoA1 << <CudaBlckNum, CudaThrdNum >> > (d_NumNodes, d_MyElem, d_MyNode, d_gamma1);

		CopyA1ToA0 << <CudaBlckNum, CudaThrdNum >> > (d_NumNodes, d_MyElem, d_MyNode);

		UpdateVe << <CudaBlckNum, CudaThrdNum >> > (d_NumElem, d_MyElem, d_MyNode);

		hipMemcpy(&PorbeValueVsItration[RelaxCount], &(d_MyNode[probeID].A0), sizeof(double), hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\nElapsed time for NDDR Iteration is %.4f ms\n", elapsedTime);
	hipMemcpy(&MyElem, d_MyElem, NumElem * sizeof(FEMElem), hipMemcpyDeviceToHost);
	hipMemcpy(&MyNode, d_MyNode, NumNodes * sizeof(FEMNode), hipMemcpyDeviceToHost);
	GPUFree();

	printf("\n%e\n", MyNode[probeID].A0);
	//for the test
	FILE *write_ptr;
	write_ptr = fopen("PorbeValueVsItration.bin", "wb");  // w for write, b for binary
	fwrite(PorbeValueVsItration, sizeof(PorbeValueVsItration), 1, write_ptr);
	//for the test

	return 0;
}


