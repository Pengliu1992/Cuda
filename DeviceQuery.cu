#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#define GET_TIME(now){ \
	double t; \
	t=clock(); \
	now =t/CLOCKS_PER_SEC ;\
}
#define N 40000
__global__ void TreeSum(int*c, int*d);
__global__ void Multiply(int*a, int* b, int* c);
__global__ void LoopSum(int*c, int*d);
void printDevProp(hipDeviceProp_t devProp);
int main()
{
	// Number of CUDA devices
	int devCount;
	double T_start, T_end;;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query...\n");
	printf("There are %d CUDA devices.\n", devCount);

	// Iterate through devices
	for (int i = 0; i < devCount; ++i)

	{
		// Get device properties
		printf("\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printDevProp(devProp);
	}
	hipSetDevice(0);
	int i, a[N], b[N], c[N], d;
	int *dev_a, *dev_b, *dev_c, *dev_d;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	hipMalloc((void**)&dev_d, 1 * sizeof(int));
	for (i = 0; i<N; i++)
	{
		a[i] = i;
		b[i] = 1;
	}
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_d, &d,  sizeof(int), hipMemcpyHostToDevice);

	Multiply << <200, 200 >> >(dev_a, dev_b, dev_c);


	GET_TIME(T_start);

	TreeSum << <1, 1 >> >(dev_c, dev_d);
	hipDeviceSynchronize();

	GET_TIME(T_end);

	hipMemcpy(&d, dev_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nResult of TreeAdder is %d\n", d);
	printf("Elapsed time for TreeAdder is %.4f s\n", (T_end - T_start));



	GET_TIME(T_start);

	LoopSum << <1, 1>> >(dev_c, dev_d);
	hipDeviceSynchronize();

	GET_TIME(T_end);
	hipMemcpy(&d, dev_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nResult of LoopAdder is %d\n", d);
	printf("Elapsed time for LoopAdder is %.4f s\n", (T_end - T_start));
	

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);

	return 0;
}

__global__ void Multiply(int*a, int* b, int* c)
{
	int bid = threadIdx.x + blockIdx.x*blockDim.x;
	if (bid<N)
		c[bid] = a[bid] * b[bid];
}

__global__ void LoopSum(int*c, int*d)
{
	int i = 0;
	int temp = 0;
	for (i = 0; i < N; i++)
		temp += c[i];

	*d = temp;
}

__global__ void TreeSum(int*c, int*d)
{
	int i = 0;
	int temp = 0;
	for (i = 0; i < N; i++)
		temp += c[i];

	*d = temp;
}

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
	printf("Major revision number:         %d\n", devProp.major);
	printf("Minor revision number:         %d\n", devProp.minor);
	printf("Name:                          %s\n", devProp.name);
	printf("Total global memory:           %u\n", devProp.totalGlobalMem);
	printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Maximum memory pitch:          %u\n", devProp.memPitch);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n", devProp.clockRate);
	printf("Total constant memory:         %u\n", devProp.totalConstMem);
	printf("Texture alignment:             %u\n", devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}