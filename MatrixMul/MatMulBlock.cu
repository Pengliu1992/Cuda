
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void MatrixMut(int *A, int *B, int *C, int N)
{

int i=blockDim.x*blockIdx.x+threadIdx.x;

int j=blockDim.y*blockIdx.y+threadIdx.y;

int temp=0;


for(int k=0;k<N;k++)
temp+= A[i*N+k]*B[k*N+j];



C[i*N+j]=temp;

}


int main()
{

const int dim=1<<10;
const int size=dim*dim;

int *A= (int *)malloc(size*sizeof(int));
int *B= (int *)malloc(size*sizeof(int));
int *C= (int *)malloc(size*sizeof(int));


for (int i = 0; i < size; ++i)
{
	
	A[i]=1;
	B[i]=2;

}


int *d_A=NULL;
int *d_B=NULL;
int *d_C=NULL;

hipMalloc((void**) &d_A,size*sizeof(int));
hipMalloc((void**) &d_B,size*sizeof(int));
hipMalloc((void**) &d_C,size*sizeof(int));



hipMemcpy(d_A,A,size*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_B,B,size*sizeof(int),hipMemcpyHostToDevice);




dim3 grid_size(64,64);
dim3 block_size(16,16);

hipEvent_t start, stop;


hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);


MatrixMut<<<grid_size,block_size>>>(d_A,d_B,d_C,dim);


hipEventRecord(stop, 0);
hipEventSynchronize(stop);

float msecTotal = 0.0f;
hipEventElapsedTime(&msecTotal, start, stop);

printf("Eclapsed time is %f ms \n", msecTotal);

hipMemcpy(C,d_C,size*sizeof(int),hipMemcpyDeviceToHost);




for (int i = 0; i < size; ++i)
{
	
	if(C[i]!=dim*2)
	{
		printf("Test Failed!");
		exit(-1);
	}

}

printf("Test Passed \n");

free(A);
free(B);
free(C);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

return 0;

}



