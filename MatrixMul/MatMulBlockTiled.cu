
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void MatrixMut(int *A, int *B, int *C, int N)
{

	__shared__ int As[256];
	__shared__ int Bs[256];


int tx=threadIdx.x, ty=threadIdx.y;

int row=blockDim.x*blockIdx.x+threadIdx.x;

int col=blockDim.y*blockIdx.y+threadIdx.y;


int temp=0;
// 16*16 subblock every time, 64 times, for each time
// each thread get 1 element, a block get 256 elements, then sysnchronize
// loop 16 multiply and summation, complete 1/64. Sync for next subblock
for(int k=0;k<64;k++)
{
	As[tx*16+ty]=A[row*N+k*16+ty];
	Bs[tx*16+ty]=B[col+(k*16+tx)*N];
	__syncthreads();

    for(int x=0;x<16;x++)
    temp+=As[tx*16+x]*Bs[x*16+ty];
	__syncthreads();
}


C[row*N+col]=temp;

}


int main()
{

const int dim=1<<10;
const int size=dim*dim;

int *A= (int *)malloc(size*sizeof(int));
int *B= (int *)malloc(size*sizeof(int));
int *C= (int *)malloc(size*sizeof(int));


for (int i = 0; i < size; ++i)
{
	
	A[i]=1;
	B[i]=2;

}


int *d_A=NULL;
int *d_B=NULL;
int *d_C=NULL;

hipMalloc((void**) &d_A,size*sizeof(int));
hipMalloc((void**) &d_B,size*sizeof(int));
hipMalloc((void**) &d_C,size*sizeof(int));



hipMemcpy(d_A,A,size*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_B,B,size*sizeof(int),hipMemcpyHostToDevice);




dim3 grid_size(64,64);
dim3 block_size(16,16);

hipEvent_t start, stop;


hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);


MatrixMut<<<grid_size,block_size>>>(d_A,d_B,d_C,dim);


hipEventRecord(stop, 0);
hipEventSynchronize(stop);

float msecTotal = 0.0f;
hipEventElapsedTime(&msecTotal, start, stop);

printf("Eclapsed time is %f ms \n", msecTotal);

hipMemcpy(C,d_C,size*sizeof(int),hipMemcpyDeviceToHost);




for (int i = 0; i < size; ++i)
{
	
	if(C[i]!=dim*2)
	{
		printf("Test Failed!\n");
		exit(-1);
	}

}

printf("Test Passed \n");

free(A);
free(B);
free(C);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

return 0;

}



