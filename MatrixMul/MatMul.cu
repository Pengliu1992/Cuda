
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void MatrixMut(int *A, int *B, int *C, int N)
{

int i=threadIdx.x;
int j=blockIdx.x;
int temp=0;


for(int k=0;k<N;k++)
temp+= A[i*N+k]*B[k*N+j];



C[i*N+j]=temp;

}


int main()
{

const int dim=1<<10;
const int size=dim*dim;

int *A= (int *)malloc(size*sizeof(int));
int *B= (int *)malloc(size*sizeof(int));
int *C= (int *)malloc(size*sizeof(int));


for (int i = 0; i < size; ++i)
{
	
	A[i]=1;
	B[i]=2;

}


int *d_A=NULL;
int *d_B=NULL;
int *d_C=NULL;

hipMalloc((void**) &d_A,size*sizeof(int));
hipMalloc((void**) &d_B,size*sizeof(int));
hipMalloc((void**) &d_C,size*sizeof(int));



hipMemcpy(d_A,A,size*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_B,B,size*sizeof(int),hipMemcpyHostToDevice);




dim3 grid_size(dim);
dim3 block_size(dim);

hipEvent_t start, stop;


hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);


MatrixMut<<<grid_size,block_size>>>(d_A,d_B,d_C,dim);


hipEventRecord(stop, 0);
hipEventSynchronize(stop);

float msecTotal = 0.0f;
hipEventElapsedTime(&msecTotal, start, stop);

printf("Eclapsed time is %f ms \n", msecTotal);

hipMemcpy(C,d_C,size*sizeof(int),hipMemcpyDeviceToHost);




for (int i = 0; i < size; ++i)
{
	
	if(C[i]!=dim*2)
	{
		printf("A[i]= %d, B[i]=%d, C[i]=%d \n", A[i], B[i], C[i]);
		exit(-1);
	}

}

printf("Test Passed \n");

free(A);
free(B);
free(C);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

return 0;

}



